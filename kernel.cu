#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>


static uint8_t* buffer=0;
static double* vecl=0;
static uint8_t* stars = 0;
static bool* blocks1 = 0;
static bool* blocks2 = 0;

const static int starsize = 100;

__device__ int collcolor2(bool* blocks, int collblock, int tmpx, int tmpy, bool rem, double coord1, double coord2, double d1,double d2,int face, bool other, int currx, int curry, int currz)
{
	int i;
	double r, g, b;
	int contid;
	double fact;

	if (rem && tmpx == 1920 / 2 && tmpy == 1080 / 2) blocks[collblock] = false;

	for (i = 0; i < 3; i++) collblock = (60493 * collblock + 11) % 115249;

	if (other)
	{
		collblock = (60493 * collblock + 11) % 115249;

		if (currz == 0)
		{
			if (collblock % 3 == 0)
			{
				r = 255.0;
				g = 0.0;
				b = 255.0;
			}
			else if (collblock % 3 == 1)
			{
				r = 0.0;
				g = 255.0;
				b = 255.0;
			}
			else
			{
				r = 255.0;
				g = 255.0;
				b = 255.0;
			}
		}
		else
		{
			if (collblock % 1001 == 0)
			{
				r = 128.0;
				g = 0.0;
				b = 255.0;
			}
			else if (collblock % 5 == 0)
			{
				r = 128.0;
				g = 128.0;
				b = 128.0;
			}
			else if (collblock % 5 == 1)
			{
				r = 200.0;
				g = 200.0;
				b = 200.0;
			}
			else if (collblock % 5 == 2)
			{
				r = 135.0;
				g = 42.0;
				b = 1.0;
			}
			else if (collblock % 5 == 3)
			{
				r = 64.0;
				g = 64.0;
				b = 64.0;
			}
			else
			{
				r = 83.0;
				g = 41.0;
				b = 0.0;
			}
		}
	}
	else
	{
		if (currz == 0)
		{
			if (curry < 19)
			{
				r = 0.0;
				g = 0.0;
				b = 255.0;
			}
			else if (curry < 20)
			{
				r = 0.0;
				g = 255.0;
				b = 255.0;
			}
			else if (curry < 21)
			{
				r = 254.0;
				g = 214.0;
				b = 131.0;
			}
			else if (curry < 22)
			{
				r = 0.0;
				g = 214.0;
				b = 0.0;
			}
			else if(curry<27)
			{
				r = 0.0;
				g = 64.0;
				b = 0.0;
			}
			else if (curry < 28)
			{
				r = 0.0;
				g = 214.0;
				b = 0.0;
			}
			else if (curry < 29)
			{
				r = 254.0;
				g = 214.0;
				b = 131.0;
			}
			else if (curry < 30)
			{
				r = 0.0;
				g = 255.0;
				b = 255.0;
			}
		}
		else
		{
			if (collblock % 1001 == 0)
			{
				r = 128.0;
				g = 0.0;
				b = 255.0;
			}
			else if (collblock % 5 == 0)
			{
				r = 128.0;
				g = 128.0;
				b = 128.0;
			}
			else if (collblock % 5 == 1)
			{
				r = 200.0;
				g = 200.0;
				b = 200.0;
			}
			else if (collblock % 5 == 2)
			{
				r = 135.0;
				g = 42.0;
				b = 1.0;
			}
			else if (collblock % 5 == 3)
			{
				r = 64.0;
				g = 64.0;
				b = 64.0;
			}
			else
			{
				r = 83.0;
				g = 41.0;
				b = 0.0;
			}
		}
	}

	contid = floor(10.0*coord1/d1)+10* floor(10.0 * coord2 / d2);

	for (i = 0; i < 3; i++) contid = (60493 * contid + 11) % 115249;
	if (contid < 0) contid += 115249;
	fact = 0.3 * contid / 115248.0;

		r =  (1.0 - fact) * r;
		g =  (1.0 - fact) * g;
		b =  (1.0 - fact) * b;


	fact = face * 0.17;

	r = (1.0 - fact) * r;
	g = (1.0 - fact) * g;
	b = (1.0 - fact) * b;

	return (int)r+256*(int)g + 256*256*(int)b;
}

__device__ int raymarch1(bool* blocks1, double rayon, double alpha, double dx, double dy, double dz, int currx, int curry, int currz, double vecn0, double vecn1, double vl, double pos0, double pos1, int nbx, int nby, double kappa, int tmpx, int tmpy, bool rem, bool other)
{
	double contx, conty, contz;
	double coordx, coordy, coordz;
	int height = floor((rayon - alpha) / dz);

	double nextcoordz = alpha + currz * dz;
	int nbxnby = nbx * nby;

	int signx = (-2) * signbit(vecn0) + 1;
	int signy = (-2) * signbit(vecn1) + 1;

	vecn0 *= signx;
	vecn1 *= signy;

	int currblock = currx + nbx * curry + nbxnby * currz;

	if (blocks1[currblock])
	{
		coordx = pos0;
		coordy = pos1;

		coordx = fmod(coordx, dx);
		coordy = fmod(coordy, dy);

		if (coordx < 0) coordx += dx;
		if (coordy < 0) coordy += dy;

		return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordy, dx, dy, 0, other, currx, curry, currz);
	}

	double t0dx = dx * vl / vecn0;
	double t0dy = dy * vl / vecn1;

	double sigx = fmod(pos0,dx);
	double sigy = fmod(pos1, dy);

	int face;
	
	if (sigx < 0) sigx += dx;
	if (sigy < 0) sigy += dy;

	if (signx == 1) contx = t0dx- sigx * vl / vecn0;
	else contx = sigx*vl/vecn0;

	if (signy == 1) conty = t0dy - sigy * vl / vecn1;
	else conty = sigy * vl / vecn1;

	
	
	if (currz < height)
	{
		nextcoordz += dz;
		contz = kappa - sqrt(rayon * rayon - nextcoordz * nextcoordz);
		
		while (true)
		{
			if (contz < contx)
			{
				if (contz < conty)
				{
					currz++;
					currblock += nbxnby;
					if (blocks1[currblock])
					{
						coordx = pos0 + contz * vecn0 *signx / vl;
						coordy = pos1 + contz * vecn1 * signy / vl;

						coordx = fmod(coordx, dx);
						coordy = fmod(coordy, dy);

						if (coordx < 0) coordx += dx;
						if (coordy < 0) coordy += dy;

						return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordy, dx,dy,0, other, currx, curry, currz);
					}
					if (currz == height)
					{
						contz = kappa + sqrt(rayon * rayon - nextcoordz * nextcoordz);
						break;
					}

					nextcoordz += dz;
					contz = kappa - sqrt(rayon * rayon - nextcoordz * nextcoordz);
					
				}
				else
				{
					curry += signy;
					curry %= nby;
					if (curry < 0) curry += nby;

					currblock = currx + nbx * curry + nbxnby * currz;
					if (blocks1[currblock])
					{
						coordx = pos0 + conty * vecn0 * signx / vl;
						coordx = fmod(coordx, dx);
						if (coordx < 0) coordx += dx;

						coordz = sqrt(rayon * rayon - (conty-kappa) * (conty - kappa));
						coordz -= alpha;
						coordz = fmod(coordz, dz);

						if (signy == 1) face = 4;
						else face = 2;

						return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
					}
					conty += t0dy;
				}
			}
			else
			{
				if (contx < conty)
				{
					currx += signx;
					currx %= nbx;
					if (currx < 0) currx += nbx;

					currblock = currx + nbx * curry + nbxnby * currz;
					if (blocks1[currblock])
					{
						coordy = pos1 + contx * vecn1 * signy / vl;
						coordy = fmod(coordy, dy);
						if (coordy < 0) coordy += dy;

						coordz = sqrt(rayon * rayon - (contx - kappa) * (contx - kappa));
						coordz -= alpha;
						coordz = fmod(coordz, dz);

						if (signx == 1) face = 3;
						else face = 1;

						return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordy, coordz, dy, dz, face, other, currx, curry, currz);
					}
					contx += t0dx;
				}
				else
				{
					curry += signy;
					curry %= nby;
					if (curry < 0) curry += nby;

					currblock = currx + nbx * curry + nbxnby * currz;
					if (blocks1[currblock])
					{
						coordx = pos0 + conty * vecn0 * signx / vl;
						coordx = fmod(coordx, dx);
						if (coordx < 0) coordx += dx;

						coordz = sqrt(rayon * rayon - (conty - kappa) * (conty - kappa));
						coordz -= alpha;
						coordz = fmod(coordz, dz);

						if (signy == 1) face = 4;
						else face = 2;

						return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
					}
					conty += t0dy;
				}
			}
		}
	}
	else contz = kappa + sqrt(rayon * rayon - nextcoordz * nextcoordz);

	while (true)
	{
		if (contz < contx)
		{
			if (contz < conty)
			{
				if (currz == 0) return -1;
				currz--;
				
				currblock -= nbxnby;

				if (blocks1[currblock])
				{
					coordx = pos0 + contz * vecn0 * signx / vl;
					coordy = pos1 + contz * vecn1 * signy / vl;

					coordx = fmod(coordx, dx);
					coordy = fmod(coordy, dy);

					if (coordx < 0) coordx += dx;
					if (coordy < 0) coordy += dy;

					return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordy, dx, dy, 5, other, currx, curry, currz);
				}

				nextcoordz -= dz;
				contz = kappa + sqrt(rayon * rayon - nextcoordz * nextcoordz);

			}
			else
			{
				curry += signy;
				curry %= nby;
				if (curry < 0) curry += nby;

				currblock = currx + nbx * curry + nbxnby * currz;
				
				if (blocks1[currblock])
				{
					coordx = pos0 + conty * vecn0 * signx / vl;
					coordx = fmod(coordx, dx);
					if (coordx < 0) coordx += dx;

					coordz = sqrt(rayon * rayon - (conty - kappa) * (conty - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signy == 1) face = 4;
					else face = 2;

					return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
				}

				conty += t0dy;
			}
		}
		else
		{
			if (contx < conty)
			{
				currx += signx;
				currx %= nbx;
				if (currx < 0) currx += nbx;

				currblock = currx + nbx * curry + nbxnby * currz;
			
				if (blocks1[currblock])
				{
					coordy = pos1 + contx * vecn1 * signy / vl;
					coordy = fmod(coordy, dy);
					if (coordy < 0) coordy += dy;

					coordz = sqrt(rayon * rayon - (contx - kappa) * (contx - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signx == 1) face = 3;
					else face = 1;

					return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordy, coordz, dy, dz, face, other, currx, curry, currz);
				}

				contx += t0dx;
			}
			else
			{
				curry += signy;
				curry %= nby;
				if (curry < 0) curry += nby;

				currblock = currx + nbx * curry + nbxnby * currz;
				if (blocks1[currblock])
				{
					coordx = pos0 + conty * vecn0 * signx / vl;
					coordx = fmod(coordx, dx);
					if (coordx < 0) coordx += dx;

					coordz = sqrt(rayon * rayon - (conty - kappa) * (conty - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signy == 1) face = 4;
					else face = 2;

					return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
				}
				conty += t0dy;
			}
		}
	}
}

__device__ int raymarch2(bool* blocks1, double rayon, double alpha, double dx, double dy, double dz, int currx, int curry, int currz, double vecn0, double vecn1, double vl, double pos0, double pos1, int nbx, int nby, double kappa, int tmpx, int tmpy, bool rem, bool other)
{
	double contx, conty, contz;
	double coordx, coordy,coordz;
	int face;

	double nextcoordz = alpha + currz * dz;
	int nbxnby = nbx * nby;

	int signx = (-2) * signbit(vecn0) + 1;
	int signy = (-2) * signbit(vecn1) + 1;

	vecn0 *= signx;
	vecn1 *= signy;

	int currblock = currx + nbx * curry + nbxnby * currz;

	double t0dx = dx * vl / vecn0;
	double t0dy = dy * vl / vecn1;

	double sigx = fmod(pos0, dx);
	double sigy = fmod(pos1, dy);

	if (sigx < 0) sigx += dx;
	if (sigy < 0) sigy += dy;

	if (signx == 1) contx = t0dx - sigx * vl / vecn0;
	else contx = sigx * vl / vecn0;

	if (signy == 1) conty = t0dy - sigy * vl / vecn1;
	else conty = sigy * vl / vecn1;

	contz = kappa + sqrt(rayon * rayon - nextcoordz * nextcoordz);

	while (true)
	{
		if (contz < contx)
		{
			if (contz < conty)
			{
				if (currz == 0) return -1;
				currz--;
				currblock -= nbxnby;

				if (blocks1[currblock])
				{
					coordx = pos0 + contz * vecn0 * signx / vl;
					coordy = pos1 + contz * vecn1 * signy / vl;

					coordx = fmod(coordx, dx);
					coordy = fmod(coordy, dy);

					if (coordx < 0) coordx += dx;
					if (coordy < 0) coordy += dy;

					return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordy, dx, dy, 5, other, currx, curry, currz);
				}

				nextcoordz -= dz;
				contz = kappa + sqrt(rayon * rayon - nextcoordz * nextcoordz);

			}
			else
			{
				curry += signy;
				curry %= nby;
				if (curry < 0) curry += nby;

				currblock = currx + nbx * curry + nbxnby * currz;
				if (blocks1[currblock])
				{
					coordx = pos0 + conty * vecn0 * signx / vl;
					coordx = fmod(coordx, dx);
					if (coordx < 0) coordx += dx;

					coordz = sqrt(rayon * rayon - (conty - kappa) * (conty - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signy == 1) face = 4;
					else face = 2;

					return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
				}
				conty += t0dy;
			}
		}
		else
		{
			if (contx < conty)
			{
				currx += signx;
				currx %= nbx;
				if (currx < 0) currx += nbx;

				currblock = currx + nbx * curry + nbxnby * currz;

				if (blocks1[currblock])
				{
					coordy = pos1 + contx * vecn1 * signy / vl;
					coordy = fmod(coordy, dy);
					if (coordy < 0) coordy += dy;

					coordz = sqrt(rayon * rayon - (contx - kappa) * (contx - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signx == 1) face = 3;
					else face = 1;

					return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordy, coordz, dy, dz, face, other, currx, curry, currz);
				}

				contx += t0dx;
			}
			else
			{
				curry += signy;
				curry %= nby;
				if (curry < 0) curry += nby;

				currblock = currx + nbx * curry + nbxnby * currz;
				if (blocks1[currblock])
				{
					coordx = pos0 + conty * vecn0 * signx / vl;
					coordx = fmod(coordx, dx);
					if (coordx < 0) coordx += dx;

					coordz = sqrt(rayon * rayon - (conty - kappa) * (conty - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signy == 1) face = 4;
					else face = 2;

					return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
				}
				conty += t0dy;
			}
		}
	}
}

__device__ int raymarch3(bool* blocks1, bool* blocks2, double rayon, double alpha, double dx, double dy, double dz, int currx, int curry, int currz, double vecn0, double vecn1, double vl, double pos0, double pos1, int nbx, int nby, int nbz, double kappa, int tmpx, int tmpy, bool rem, bool other)
{
	double contx, conty, contz;
	double coordx, coordy, coordz;
	int face;
	double nextcoordz = alpha + currz * dz+dz;
	contz = kappa - sqrt(rayon * rayon - nextcoordz * nextcoordz);
	int nbxnby = nbx * nby;

	double sub = 2.0 * sqrt(rayon * rayon - (alpha + dz * nbz) * (alpha + dz * nbz));

	int signx = (-2) * signbit(vecn0) + 1;
	int signy = (-2) * signbit(vecn1) + 1;

	vecn0 *= signx;
	vecn1 *= signy;

	int currblock = currx + nbx * curry + nbxnby * currz;

	if (blocks1[currblock])
	{
		coordx = pos0;
		coordy = pos1;

		coordx = fmod(coordx, dx);
		coordy = fmod(coordy, dy);

		if (coordx < 0) coordx += dx;
		if (coordy < 0) coordy += dy;

		return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordy, dx, dy, 0, other, currx, curry, currz);
	}

	double t0dx = dx * vl / vecn0;
	double t0dy = dy * vl / vecn1;

	double sigx = fmod(pos0, dx);
	double sigy = fmod(pos1, dy);

	if (sigx < 0) sigx += dx;
	if (sigy < 0) sigy += dy;

	if (signx == 1) contx = t0dx - sigx * vl / vecn0;
	else contx = sigx * vl / vecn0;

	if (signy == 1) conty = t0dy - sigy * vl / vecn1;
	else conty = sigy * vl / vecn1;


	

		while (true)
		{
			if (contz < contx)
			{
				if (contz < conty)
				{
					if (currz == nbz - 1) break;
					currz++;
					currblock += nbxnby;
					
					if (blocks1[currblock])
					{
						coordx = pos0 + contz * vecn0 * signx / vl;
						coordy = pos1 + contz * vecn1 * signy / vl;

						coordx = fmod(coordx, dx);
						coordy = fmod(coordy, dy);

						if (coordx < 0) coordx += dx;
						if (coordy < 0) coordy += dy;

						return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordy, dx, dy, 0, other, currx, curry, currz);
					}

					nextcoordz += dz;
					contz = kappa - sqrt(rayon * rayon - nextcoordz * nextcoordz);

				}
				else
				{
					curry += signy;
					curry %= nby;
					if (curry < 0) curry += nby;

					currblock = currx + nbx * curry + nbxnby * currz;

					if (blocks1[currblock])
					{
						coordx = pos0 + conty * vecn0 * signx / vl;
						coordx = fmod(coordx, dx);
						if (coordx < 0) coordx += dx;

						coordz = sqrt(rayon * rayon - (conty - kappa) * (conty - kappa));
						coordz -= alpha;
						coordz = fmod(coordz, dz);

						if (signy == 1) face = 4;
						else face = 2;

						return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
					}

					conty += t0dy;
				}
			}
			else
			{
				if (contx < conty)
				{
					currx += signx;
					currx %= nbx;
					if (currx < 0) currx += nbx;

					currblock = currx + nbx * curry + nbxnby * currz;
					
					if (blocks1[currblock])
					{
						coordy = pos1 + contx * vecn1 * signy / vl;
						coordy = fmod(coordy, dy);
						if (coordy < 0) coordy += dy;

						coordz = sqrt(rayon * rayon - (contx - kappa) * (contx - kappa));
						coordz -= alpha;
						coordz = fmod(coordz, dz);

						if (signx == 1) face = 3;
						else face = 1;

						return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordy, coordz, dy, dz, face, other, currx, curry, currz);
					}

					contx += t0dx;
				}
				else
				{
					curry += signy;
					curry %= nby;
					if (curry < 0) curry += nby;

					currblock = currx + nbx * curry + nbxnby * currz;
					
					if (blocks1[currblock])
					{
						coordx = pos0 + conty * vecn0 * signx / vl;
						coordx = fmod(coordx, dx);
						if (coordx < 0) coordx += dx;

						coordz = sqrt(rayon * rayon - (conty - kappa) * (conty - kappa));
						coordz -= alpha;
						coordz = fmod(coordz, dz);

						if (signy == 1) face = 4;
						else face = 2;

						return collcolor2(blocks1, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
					}

					conty += t0dy;
				}
			}
		}

		other = !other;

		if (blocks2[currblock])
		{
			coordx = pos0 + contz * vecn0 * signx / vl;
			coordy = pos1 + contz * vecn1 * signy / vl;

			coordx = fmod(coordx, dx);
			coordy = fmod(coordy, dy);

			if (coordx < 0) coordx += dx;
			if (coordy < 0) coordy += dy;

			return collcolor2(blocks2, currblock, tmpx, tmpy, rem, coordx, coordy, dx, dy, 5, other, currx, curry, currz);
		}

		nextcoordz -= dz;
		contz = kappa + sqrt(rayon * rayon - nextcoordz * nextcoordz) - sub;

	while (true)
	{
		if (contz < contx)
		{
			if (contz < conty)
			{
				if (currz == 0) return -1;
				currz--;

				currblock -= nbxnby;

				if (blocks2[currblock])
				{
					coordx = pos0 + contz * vecn0 * signx / vl;
					coordy = pos1 + contz * vecn1 * signy / vl;

					coordx = fmod(coordx, dx);
					coordy = fmod(coordy, dy);

					if (coordx < 0) coordx += dx;
					if (coordy < 0) coordy += dy;

					return collcolor2(blocks2, currblock, tmpx, tmpy, rem, coordx, coordy, dx, dy, 5, other, currx, curry, currz);
				}


				nextcoordz -= dz;
				contz = kappa + sqrt(rayon * rayon - nextcoordz * nextcoordz)-sub;

			}
			else
			{
				curry += signy;
				curry %= nby;
				if (curry < 0) curry += nby;

				currblock = currx + nbx * curry + nbxnby * currz;
				
				if (blocks2[currblock])
				{
					coordx = pos0 + conty * vecn0 * signx / vl;
					coordx = fmod(coordx, dx);
					if (coordx < 0) coordx += dx;

					coordz = sqrt(rayon * rayon - (conty +sub - kappa) * (conty +sub - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signy == 1) face = 4;
					else face = 2;

					return collcolor2(blocks2, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
				}

				conty += t0dy;
			}
		}
		else
		{
			if (contx < conty)
			{
				currx += signx;
				currx %= nbx;
				if (currx < 0) currx += nbx;

				currblock = currx + nbx * curry + nbxnby * currz;
				
				if (blocks2[currblock])
				{
					coordy = pos1 + contx * vecn1 * signy / vl;
					coordy = fmod(coordy, dy);
					if (coordy < 0) coordy += dy;

					coordz = sqrt(rayon * rayon - (contx +sub - kappa) * (contx +sub - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signx == 1) face = 3;
					else face = 1;

					return collcolor2(blocks2, currblock, tmpx, tmpy, rem, coordy, coordz, dy, dz, face, other, currx, curry, currz);
				}

				contx += t0dx;
			}
			else
			{
				curry += signy;
				curry %= nby;
				if (curry < 0) curry += nby;

				currblock = currx + nbx * curry + nbxnby * currz;

				if (blocks2[currblock])
				{
					coordx = pos0 + conty * vecn0 * signx / vl;
					coordx = fmod(coordx, dx);
					if (coordx < 0) coordx += dx;

					coordz = sqrt(rayon * rayon - (conty + sub - kappa) * (conty + sub - kappa));
					coordz -= alpha;
					coordz = fmod(coordz, dz);

					if (signy == 1) face = 4;
					else face = 2;

					return collcolor2(blocks2, currblock, tmpx, tmpy, rem, coordx, coordz, dx, dz, face, other, currx, curry, currz);
				}

				conty += t0dy;
			}
		}
	}
}

__device__ void tormat(double phi, double theta, double* mat)
{
	mat[0] = cos(theta) * sin(phi);
	mat[3] = cos(theta) * cos(phi);
	mat[6] = sin(theta);

	mat[2] = cos(phi);
	mat[5] = -sin(phi);
	mat[8] = 0;

	mat[1] = -sin(theta) * sin(phi);
	mat[4] = -sin(theta) * cos(phi);
	mat[7] = cos(theta);
}

__device__ double matdet(double* m)
{
	return m[0] * (m[4]*m[8]-m[5]*m[7]) - m[1] * (m[3]*m[8]-m[5]*m[6]) + m[2] * (m[3]*m[7]-m[4]*m[6]);
}

__device__ void matinv(double* m, double* res)
{
	res[0] = m[4] * m[8] - m[5] * m[7];
	res[1] = m[2] * m[7] - m[1] * m[8];
	res[2] = m[1] * m[5] - m[2] * m[4];
	res[3] = m[5] * m[6] - m[3] * m[8];
	res[4] = m[0] * m[8] - m[2] * m[6];
	res[6] = m[3] * m[7] - m[4] * m[6];
	res[5] = m[2] * m[3] - m[0] * m[5];
	res[7] = m[1] * m[6] - m[0] * m[7];
	res[8] = m[0] * m[4] - m[1] * m[3];
}

__device__ void matmult(double* m1, double* m2, double* res)
{
	res[0] = m1[0] * m2[0] + m1[1] * m2[3] + m1[2] * m2[6];
	res[1] = m1[0] * m2[1] + m1[1] * m2[4] + m1[2] * m2[7];
	res[2] = m1[0] * m2[2] + m1[1] * m2[5] + m1[2] * m2[8];
	res[3] = m1[3] * m2[0] + m1[4] * m2[3] + m1[5] * m2[6];
	res[4] = m1[3] * m2[1] + m1[4] * m2[4] + m1[5] * m2[7];
	res[5] = m1[3] * m2[2] + m1[4] * m2[5] + m1[5] * m2[8];
	res[6] = m1[6] * m2[0] + m1[7] * m2[3] + m1[8] * m2[6];
	res[7] = m1[6] * m2[1] + m1[7] * m2[4] + m1[8] * m2[7];
	res[8] = m1[6] * m2[2] + m1[7] * m2[5] + m1[8] * m2[8];
}

__device__ void matact(double* m, double vecn0, double vecn1, double vecn2, double* nvecn)
{
	nvecn[0] = m[0] * vecn0 + m[1] * vecn1 + m[2] * vecn2;
	nvecn[1] = m[3] * vecn0 + m[4] * vecn1 + m[5] * vecn2;
	nvecn[2] = m[6] * vecn0 + m[7] * vecn1 + m[8] * vecn2;
}

__device__ void matflip(double* m, double* res)
{
	res[0] = m[6];
	res[1] = m[7];
	res[2] = m[8];
	res[3] = m[3];
	res[4] = m[4];
	res[5] = m[5];
	res[6] = -m[0];
	res[7] = -m[1];
	res[8] = -m[2];
}

__device__ void matflip2(double* m, double* res)
{
	res[0] = m[2];
	res[1] = m[1];
	res[2] = -m[0];
	res[3] = m[5];
	res[4] = m[4];
	res[5] = -m[3];
	res[6] = m[8];
	res[7] = m[7];
	res[8] = -m[6];
}

__device__ double solvequartic(double a0, double b0, double c0, double d0, double e0)
{
	double tmp;
	double tmin = 65536.0;
	double sint,s;
	double r1, qds, rootint;

	double b = b0 / a0;
	double c = c0 / a0;
	double d = d0 / a0;
	double e = e0 / a0;

	double c2 = c * c;
	double bd = b * d;
	double c3 = c2 * c;
	double bcd = bd * c;
	double b2 = b * b;
	double b2e = b2 * e;
	double d2 = d * d;
	double ce = c * e;
	double bc = b * c;
	double b3 = b2 * b;
	double mbd4 = (-0.25) * b;

	double t0 = c2 - 3.0 * bd + 12.0 * e;
	double t1 = 2.0 * c3 - 9.0 * bcd + 27.0 * b2e + 27.0 * d2 - 72.0 * ce;
	double p = (8.0 * c - 3.0 * b2) / 8.0;
	double q = (b3 - 4.0 * bc + 8.0 * d) / 8.0;

	double disc = t1 * t1 - 4.0 * t0 * t0 * t0;
	
	if (disc < 0)
	{
		double st0 = sqrt(t0);
		double phi = (acos(t1 / (2.0 * t0 * st0))) / 3.0;
		sint = (-2.0 / 3.0) * p + (2.0 / 3.0) * st0 * cos(phi);
	}
	else
	{
		double bigq = cbrt((t1 + sqrt(disc)) * 0.5);
		sint = (-2.0 / 3.0) * p + (1.0 / 3.0) * (bigq + t0 / bigq);
	}	

	s = sqrt(sint) * 0.5;
	
	rootint = (sint + 2.0 * p) * (-1.0);
	qds = q / s;

	r1 = rootint + qds;

	if (r1 > 0)
	{
		r1 = 0.5 * sqrt(r1);
		tmp = mbd4 - s;

		if (tmp + r1 > 0.0000001 && tmp + r1 < tmin) tmin = tmp + r1;
		if (tmp - r1 > 0.0000001 && tmp - r1 < tmin) tmin = tmp - r1;
	}

	r1 = rootint - qds;

	if (r1 > 0)
	{
		r1 = 0.5 * sqrt(r1);
		tmp = mbd4 + s;

		if (tmp + r1 > 0.0000001 && tmp + r1 < tmin) tmin = tmp + r1;
		if (tmp - r1 > 0.0000001 && tmp - r1 < tmin) tmin = tmp - r1;
	}

	return tmin;
}

__device__ double toruscoll(double a, double b, double c, double d, double e, double f, double m, double n)
{
	double t4, t3, t2, t1, t0;

	double a2 = a * a;
	double b2 = b * b;
	double c2 = c * c;
	double d2 = d * d;
	double e2 = e * e;
	double f2 = f * f;

	double ab = a * b;
	double cd = c * d;
	double ef = e * f;

	double sum1 = a2 + c2 + e2;
	double sum2 = ab + cd;
	double sum3 = sum2 + ef;
	double sum4 = m + n;
	double sum5 = b2 + d2 + f2;
	double sum6 = m - n;
	double sum7 = ab + ef;

	t0 = sum5 * sum5 + sum6 * sum6;
	t0 += (-2.0) * (sum5 * sum4 - 2.0 * f2 * n);

	t1 = (b2 + d2 + f2) * sum3;
	t1 -= sum3 * sum4;
	t1 += 2.0 * ef * n;
	t1 *= 4.0;

	t2 = d * (d * (sum1 + 2.0 * c2) + 4.0 * c * sum7) + b * (b * (sum1 + 2.0 * a2) + 4.0 * a * ef) + f2 * (sum1 + 2.0 * e2);
	t2 -= sum1 * sum4;
	t2 += 2.0 * e2 * n;
	t2 *= 2.0;

	t3 = 4.0 * sum1 * sum3;

	t4 = sum1 * sum1;

	return solvequartic(t4, t3, t2, t1, t0);
}

__global__ void bufferinit(uint8_t* buffer)
{
	buffer[4 * (blockIdx.x * blockDim.x + threadIdx.x) + 3] = 255;
}

__global__ void setstars(uint8_t* stars)
{
	int i;
	int tmp = blockIdx.x * blockDim.x + threadIdx.x;

	int rand = tmp;

	for (i = 0; i < 10; i++) rand = (60493 * rand + 11) % 115249;

	if ((rand) % 5 == 0)
	{
		stars[tmp] = 255 * rand / 115249;
	}
	else
	{
		stars[tmp] = 0;
	}
}

__global__ void setblocks(bool* blocks1, bool* blocks2)
{
	for (int i = 0; i < 30 * 30 * 30; i++)
	{
		blocks1[i] = true;
		blocks2[i] = true;
	}

}

__global__ void setblocksrand(bool* blocks1, bool* blocks2)
{
	int i;
	int rand=1;

	for (i = 0; i < 30 * 30 * 30; i++)
	{
		rand = (60493 * rand + 11) % 115249;
		if (rand % 100 == 0) blocks1[i] = true;
		else blocks1[i] = false;
		rand = (60493 * rand + 11) % 115249;
		if (rand % 100 == 0) blocks2[i] = true;
		else blocks2[i] = false;
	}

}

__global__ void veclinit(double* vecl, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int tmpx = i % 1920;
	int tmpy = (i - tmpx) / 1920;

	double vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
	double vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
	double vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

	vecl[i] = sqrt(vecn0 * vecn0 + vecn1 * vecn1 + vecn2 * vecn2);
}

__global__ void addKernel(uint8_t* buffer, double* vecl, double pos0,double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, bool inside, double alpha, double beta, double bigr, double r, bool other, uint8_t* stars, bool* blocks1, bool* blocks2, double dx, double dy, double dz, int currx, int curry, int currz, int nbx, int nby,int nbz, bool rem)
{
	int i;
	double vecn0, vecn1, vecn2;

	double geoang;
	double xyvec;

	double inv[9]{};
	double nvecn[3]{};
	double npos[3]{};
	double vl;

	double torcoll;
	double tor0, tor1, tor2;
	double theta, phi;

	double exitalpha;

	double rayon;
	double kappa;
	double exit;

	int collrgb;
	double skyr, skyg, skyb;

	double kesum;


	double u, v;
	uint8_t uv;

	int tmp2;
	int tmp = blockIdx.x * blockDim.x + threadIdx.x;
	int tmpx = tmp % 1920;
	int tmpy = (tmp-tmpx) /1920;

	double mat1[9]{};
	/*
	__shared__ bool sblocks1[30 * 30 * 30];
	__shared__ bool sblocks2[30 * 30 * 30];

	tmp2 = (30 * 30 * 30) / 600;

	for (i = 0; i < tmp2; i++)
	{
		sblocks1[threadIdx.x * tmp2 + i] = blocks1[threadIdx.x * tmp2 + i];
		sblocks2[threadIdx.x * tmp2 + i] = blocks2[threadIdx.x * tmp2 + i];
	}
	__syncthreads();
	*/
	
	vecn0 = vec0 + tmpx * addy0 + tmpy * addz0;
	vecn1 = vec1 + tmpx * addy1 + tmpy * addz1;
	vecn2 = vec2 + tmpx * addy2 + tmpy * addz2;

	vecn0 /= vecl[tmp];
	vecn1 /= vecl[tmp];
	vecn2 /= vecl[tmp];

	if (inside)
	{
		vl = sqrt(1.0 - vecn2 * vecn2);
		geoang = atan(vecn2 / vl);
		rayon = pos2 / cos(geoang);
		kappa = sin(geoang) * rayon;
		exitalpha = sqrt(rayon * rayon - alpha * alpha);
		kesum = kappa + exitalpha;

		if(kappa>0)
		{
			if (beta < rayon)
			{
				if (other) collrgb = raymarch3(blocks2, blocks1, rayon, alpha, dx, dy, dz, currx, curry, currz, vecn0, vecn1, vl, pos0, pos1, nbx, nby, nbz,kappa, tmpx, tmpy, rem, other);
				else collrgb = raymarch3(blocks1, blocks2, rayon, alpha, dx, dy, dz, currx, curry, currz, vecn0, vecn1, vl, pos0, pos1, nbx, nby,nbz, kappa, tmpx, tmpy, rem, other);

				if (collrgb != -1)
				{
					buffer[4 * tmp] = collrgb % 256;
					collrgb -= collrgb % 256;
					collrgb /= 256;
					buffer[4 * tmp + 1] = collrgb % 256;
					collrgb -= collrgb % 256;
					collrgb /= 256;
					buffer[4 * tmp + 2] = collrgb % 256;
					return;
				}

				exit = sqrt(rayon * rayon - beta * beta);
				kesum -= 2.0 * exit;
				other = !other;
			}
			else
			{
				if(other) collrgb=raymarch1(blocks2,rayon, alpha, dx, dy, dz, currx, curry, currz, vecn0, vecn1, vl, pos0, pos1, nbx, nby, kappa, tmpx, tmpy, rem,true);
				else collrgb = raymarch1(blocks1, rayon, alpha, dx, dy, dz, currx, curry, currz, vecn0, vecn1, vl, pos0, pos1, nbx, nby, kappa, tmpx, tmpy, rem,false);

				if (collrgb != -1)
				{
					buffer[4 * tmp] = collrgb%256;
					collrgb-= collrgb % 256;
					collrgb /= 256;
					buffer[4 * tmp + 1] = collrgb % 256;
					collrgb -= collrgb % 256;
					collrgb /= 256;
					buffer[4 * tmp + 2] = collrgb % 256;
					return;
				}
			}
		}
		else
		{
			if (other) collrgb = raymarch2(blocks2, rayon, alpha, dx, dy, dz, currx, curry, currz, vecn0, vecn1, vl, pos0, pos1, nbx, nby, kappa, tmpx,  tmpy, rem, other);
			else collrgb = raymarch2(blocks1, rayon, alpha, dx, dy, dz, currx, curry, currz, vecn0, vecn1, vl, pos0, pos1, nbx, nby, kappa, tmpx, tmpy, rem, other);

			if (collrgb != -1)
			{
				buffer[4 * tmp] = collrgb % 256;
				collrgb -= collrgb % 256;
				collrgb /= 256;
				buffer[4 * tmp + 1] = collrgb % 256;
				collrgb -= collrgb % 256;
				collrgb /= 256;
				buffer[4 * tmp + 2] = collrgb % 256;
				return;
			}
		}

		

		pos0 += kesum * vecn0 / vl;
		pos1 += kesum * vecn1 / vl;

		vecn0 = sqrt(1.0 - (exitalpha * exitalpha) / (rayon * rayon)) * vecn0 / vl;
		vecn1 = sqrt(1.0 - (exitalpha * exitalpha) / (rayon * rayon)) * vecn1 / vl;
		vecn2 = -exitalpha / rayon;

		pos0 /= bigr;
		pos1 /= r;

		npos[0] = sin(pos0) * (bigr + r * cos(pos1));
		npos[1] = cos(pos0) * (bigr + r * cos(pos1));
		npos[2] = r * sin(pos1);

		tormat(pos0, pos1, mat1);
		matflip2(mat1, inv);
		matact(inv, vecn0, vecn1, vecn2, nvecn);

		vecn0 = nvecn[0];
		vecn1 = nvecn[1];
		vecn2 = nvecn[2];

		pos0 = npos[0];
		pos1 = npos[1];
		pos2 = npos[2];
	}


	torcoll = toruscoll(vecn0, pos0, vecn1, pos1, vecn2, pos2, r*r, bigr*bigr);

	for (i = 0; i < 10; i++) {
		if (torcoll != 65536)
		{
			tor0 = pos0 + torcoll * vecn0;
			tor1 = pos1 + torcoll * vecn1;
			tor2 = pos2 + torcoll * vecn2;
			xyvec = sqrt(tor0 * tor0 + tor1 * tor1);

			if (abs(tor2 / r) > 1 || abs(tor1 / xyvec) > 1)
			{
				buffer[4 * tmp] = 0;
				buffer[4 * tmp + 1] = 0;
				buffer[4 * tmp + 2] = 0;
				return;
			}

			theta = asin(tor2 / r);
			if (xyvec < bigr) theta = M_PI - theta;
			if (theta < 0) theta += 2.0 * M_PI;

			phi = acos(tor1 / xyvec);
			if (tor0 < 0) phi *= -1.0;
			if (phi < 0) phi += 2.0 * M_PI;

			tormat(phi, theta, mat1);
			matinv(mat1, inv);
			matflip(inv, mat1);
			matact(mat1, vecn0, vecn1, vecn2, nvecn);

			npos[0] = phi * (bigr);
			npos[1] = theta * r;


			vl = sqrt(1.0 - nvecn[2] * nvecn[2]);
			geoang = atan(nvecn[2] / vl);
			rayon = alpha / cos(geoang);
			kappa = sin(geoang) * rayon;

			currx = floor(npos[0] * nbx / (2.0 * M_PI * bigr));
			curry = floor(npos[1] * nby / (2.0 * M_PI * r));

			if (beta < rayon)
			{
				if (other) collrgb = raymarch3(blocks2, blocks1, rayon, alpha, dx, dy, dz, currx, curry, 0,nvecn[0], nvecn[1], vl, npos[0], npos[1], nbx, nby, nbz,kappa, tmpx, tmpy, rem,other);
				else collrgb = raymarch3(blocks1, blocks2, rayon, alpha, dx, dy, dz, currx, curry, 0, nvecn[0], nvecn[1], vl, npos[0], npos[1], nbx, nby, nbz,kappa, tmpx, tmpy, rem,other);

				if (collrgb != -1)
				{
					buffer[4 * tmp] = collrgb % 256;
					collrgb -= collrgb % 256;
					collrgb /= 256;
					buffer[4 * tmp + 1] = collrgb % 256;
					collrgb -= collrgb % 256;
					collrgb /= 256;
					buffer[4 * tmp + 2] = collrgb % 256;
					return;
				}

				exitalpha = sqrt(rayon * rayon - alpha * alpha);
				exit = sqrt(rayon * rayon - beta * beta);

				npos[0] += (kappa - 2.0 * exit + exitalpha) * nvecn[0] / vl;
				npos[1] += (kappa - 2.0 * exit + exitalpha) * nvecn[1] / vl;

				nvecn[0] = sqrt(1.0 - (exitalpha * exitalpha) / (rayon * rayon)) * nvecn[0] / vl;
				nvecn[1] = sqrt(1.0 - (exitalpha * exitalpha) / (rayon * rayon)) * nvecn[1] / vl;
				nvecn[2] = -exitalpha / rayon;

				other = !other;
			}
			else
			{
				
					if (other) collrgb = raymarch1(blocks2, rayon, alpha, dx, dy, dz, currx, curry, 0, nvecn[0], nvecn[1], vl, npos[0], npos[1], nbx, nby, kappa, tmpx, tmpy, rem,other);
					else collrgb = raymarch1(blocks1, rayon, alpha, dx, dy, dz, currx, curry, 0, nvecn[0], nvecn[1], vl, npos[0], npos[1], nbx, nby, kappa, tmpx, tmpy, rem,other);

					if (collrgb != -1)
					{
						buffer[4 * tmp] = collrgb % 256;
						collrgb -= collrgb % 256;
						collrgb /= 256;
						buffer[4 * tmp + 1] = collrgb % 256;
						collrgb -= collrgb % 256;
						collrgb /= 256;
						buffer[4 * tmp + 2] = collrgb % 256;
						return;
					}

				npos[0] += 2.0 * kappa * nvecn[0] / vl;
				npos[1] += 2.0 * kappa * nvecn[1] / vl;

				nvecn[2] *= -1.0;
			}
			
			

			npos[0] /= bigr;
			npos[1] /= r;


			pos0 = sin(npos[0]) * (bigr + r * cos(npos[1]));
			pos1 = cos(npos[0]) * (bigr + r * cos(npos[1]));
			pos2 = r * sin(npos[1]);

			tormat(npos[0], npos[1], mat1);
			matflip2(mat1, inv);
			matact(inv, nvecn[0], nvecn[1], nvecn[2], nvecn);

			vecn0 = nvecn[0];
			vecn1 = nvecn[1];
			vecn2 = nvecn[2];

			torcoll = toruscoll(vecn0, pos0, vecn1, pos1, vecn2, pos2, r * r, bigr * bigr);
		}
	}


	if (torcoll != 65536)
	{
		buffer[4 * tmp] = 0;
		buffer[4 * tmp + 1] = 0;
		buffer[4 * tmp + 2] = 0;
		return;
	}

	if (other)
	{
		u = starsize * ((0.5 + atan2(vecn1, vecn0) / (2.0 * M_PI)));
		v = starsize * ((0.5 + asin(vecn2) / M_PI));

		tmp2 = (int)u + starsize * (int)v;
		uv = stars[tmp2];

		if (uv % 3 == 0)
		{
			buffer[4 * tmp] = 0;
			buffer[4 * tmp + 1] = 0;
			buffer[4 * tmp + 2] = (uv * uv * uv * uv) / (255.0 * 255.0 * 255.0);
		}
		else
		{
			buffer[4 * tmp] = (uv*uv*uv) / ( 255.0*255.0);
			buffer[4 * tmp + 1] = (uv*uv*uv) / ( 255.0* 255.0);
			buffer[4 * tmp + 2] = (uv*uv*uv)/( 255.0* 255.0 );
		}
	}
	else
	{
		skyr = 96.0;
		skyg = 149.0;
		skyb = 217.0;

		v = ((0.5 + asin(vecn2) / M_PI));

		buffer[4 * tmp] = v*255.0 + (1-v)*skyr;
		buffer[4 * tmp + 1] = v * 255.0 + (1 - v) * skyg;
		buffer[4 * tmp + 2] = v * 255.0 + (1 - v) * skyb;

	}
}

void cudaInit()
{
	double dist = 2.0;
	double sqsz = 0.01 / 4.0;
	double* vecltmp = new double[1920 * 1080];

	double vec0, vec1, vec2;
	double addy0, addy1, addy2;
	double addz0, addz1, addz2;
	double x00 = 1, x01 = 0, x02 = 0;
	double x10 = 0, x11 = 1, x12 = 0;
	double x20 = 0, x21 = 0, x22 = 1;
	double multy = (1 - 1920) * sqsz / 2;
	double multz = (1080 - 1) * sqsz / 2;

	hipSetDevice(0);
	hipMalloc((void**)&buffer, 4 * 1920 * 1080 * sizeof(uint8_t));
	hipMalloc((void**)&vecl, 1920 * 1080 * sizeof(double));
	hipMalloc((void**)&stars, starsize * starsize * sizeof(uint8_t));
	hipMalloc((void**)&blocks1, 30*30*30);
	hipMalloc((void**)&blocks2, 30 * 30 * 30);


	vec0 = dist * x00 + multy * x10 + multz * x20;
	vec1 = dist * x01 + multy * x11 + multz * x21;
	vec2 = dist * x02 + multy * x12 + multz * x22;
	
	addy0 = sqsz * x10;
	addy1 = sqsz * x11;
	addy2 = sqsz * x12;

	addz0 = -sqsz * x20;
	addz1 = -sqsz * x21;
	addz2 = -sqsz * x22;

	veclinit << <(int)(1920 * 1080 / 600), 600 >> > (vecl,vec0,vec1,vec2,addy0,addy1,addy2,addz0,addz1,addz2);
	hipDeviceSynchronize();

	bufferinit << <(int)(1920 * 1080 / 600), 600 >> > (buffer);
	hipDeviceSynchronize();

	setstars << <starsize * starsize / 500, 500 >> > (stars);
	hipDeviceSynchronize();

	setblocks << <1,1 >> > (blocks1,blocks2);
	hipDeviceSynchronize();
}

void cudaExit()
{
	hipFree(buffer);
	hipFree(vecl);
	hipDeviceReset();
}

void cudathingy(uint8_t* pixels, double pos0, double pos1, double pos2, double vec0, double vec1, double vec2, double addy0, double addy1, double addy2, double addz0, double addz1, double addz2, bool inside, double alpha, double beta, double bigr, double r, bool other,double dx, double dy, double dz, int currx, int curry, int currz, int nbx, int nby, int nbz, bool* cpublocks1, bool* cpublocks2, bool rem, bool blockrand, bool reset)
{
	if (blockrand) setblocksrand << <1,1 >> > (blocks1, blocks2);
	if(reset) setblocks << <1, 1 >> > (blocks1, blocks2);


	addKernel << <(int)(1920 * 1080 / 100), 100 >> > (buffer, vecl, pos0, pos1, pos2, vec0, vec1, vec2, addy0, addy1, addy2, addz0, addz1, addz2, inside, alpha, beta, bigr, r, other, stars, blocks1, blocks2, dx, dy, dz, currx, curry, currz, nbx, nby, nbz, rem);

	hipDeviceSynchronize();
	hipMemcpy(pixels, buffer, 4 * 1920 * 1080 * sizeof(uint8_t), hipMemcpyDeviceToHost);

	if (rem)
	{
		hipMemcpy(cpublocks1, blocks1, 30*30*30, hipMemcpyDeviceToHost);
		hipMemcpy(cpublocks2, blocks2, 30 * 30 * 30, hipMemcpyDeviceToHost);
	}

	
}
